#include "hip/hip_runtime.h"
//#include <bits/stdc++.h>
#include <iostream>
#include <fstream>
#include <cassert>
#include <string>
#include <sstream>
#include <vector>

#include "MyPolygon.h"


using namespace std;

#define BLOCK_SIZE 1024

__global__ void gpu_internalpoints(Point* po, Point* p, int num_vertices, bool* ips)         //(d_waits,d_Polypoints,Polycount,d_ips)
{
    int Pointindex = blockIdx.x * blockDim.x + threadIdx.x; 
    ips[Pointindex]=false;       
    int i = 0, j = num_vertices - 1;
    for (i; i < num_vertices; i++) {
        if ((po[Pointindex].y >= p[i].y) != (po[Pointindex].y >= p[j].y)) {                                      
            int linex = p[i].x + (po[Pointindex].y - p[i].y) * (p[i].x - p[j].x) / (p[i].y - p[j].y);
            if (linex > po[Pointindex].x) {
                ips[Pointindex] = !ips[Pointindex];
            }
        }
        j = i;
    }

}

string readhang(string file){
    string hang;
    ifstream infile;
    infile.open(file.data());
    assert(infile.is_open());

    getline(infile,hang);
    getline(infile,hang);
    //cout<<wkt;
    infile.close();
    return hang;
}

void stringsplit(string str,const char split,vector<string>& raw){
	istringstream iss(str);
	string token;
	while (getline(iss, token, split))
	{
		raw.push_back(token);
	}
}

void rawpointsplit(vector<string>& raw,const char split,Point *coordinate){
	string token;
	stringstream stream;
	double n;
	for(int i=0;i<raw.size();i++){
        istringstream iss(raw[i]);
        getline(iss, token, split);
        stream<<token;
        assert(n>=0);
        stream>>n;
        coordinate[i].x=n;
        stream.clear();

        getline(iss, token, split);
        stream<<token;
        stream>>n;
        assert(n>=0);
        coordinate[i].y=n;
        //coordinate[i].print();
        stream.clear();
	}

}

vector<Point > internalpoints(VertexSequence polygon,MyRaster raster){
    int dimx=raster.get_dimx();
    int dimy=raster.get_dimy();
    vector<Point > ip;
	for(double i=0;i<=dimx;i++){
		for(double j=0;j<=dimy;j++){
            Pixel *checkpix=raster.get(i,j);
            if(polygon.contain(checkpix->centralpoint)){
                ip.push_back(checkpix->centralpoint);
            }

		}
	};
	return ip;
}

void writewkt(vector<Point> ips){
    ofstream out("ips.wkt");
    if (out.is_open()){
        out << "MultiPoint(";
        for(int i=0;i<ips.size();i++){
            if(i>=1) out<<",";
            out<<"("<<ips[i].x<<" "<<ips[i].y<<")";
        }
        out<<")";
        out.close();
    }
}

int main(){
    string filename="bigpolygon.wkt";           
    string wkt=readhang(filename);
    vector<string> rawpoints;

	stringsplit(wkt,',',rawpoints);
    rawpoints[0]=rawpoints[0].substr(9,rawpoints[0].length());                      //POLYGON((129.536643 49.398036
    int last=rawpoints.size()-1;
    rawpoints[last]=rawpoints[last].substr(0,rawpoints[last].length()-2);           //129.520371 49.413440))

    Point *pp=new Point[rawpoints.size()];
    rawpointsplit(rawpoints,' ',pp);
    VertexSequence polygon(rawpoints.size(),pp);
    cout<<polygon.num_vertices<<endl;
    Point position(92.6, 43.7);
    if(polygon.contain(position)){
        cout<<"surely contain";
    }
    else cout<<"out";
    cout<<endl;
    box *MBR=polygon.getMBR();
    MBR->print();
    cout<<endl;
    double s=MBR->area();
    cout<<s<<endl;

    int num_pixel=1000000;
    MyRaster raster(&polygon,num_pixel);
    raster.init_pixels();

    int waitscount=raster.candidates.size();
    int Polycount=polygon.num_vertices;
    Point *waits, *Polypoints;
    bool *containIndex;
    hipHostMalloc((void**)&waits, sizeof(Point) * waitscount);
    hipHostMalloc((void**)&Polypoints, sizeof(Point) * Polycount);
    hipHostMalloc((void**)&containIndex, sizeof(bool) * waitscount);
    for(int i=0;i<waitscount;i++){
    	waits[i]=raster.candidates.front();
    	raster.candidates.pop();
    }
    
    Polypoints=polygon.p;
    
    Point *d_waits, *d_Polypoints;
    bool *d_ips;
    hipMalloc((void**)&d_waits, sizeof(Point) * waitscount);
    hipMalloc((void**)&d_Polypoints, sizeof(Point) * Polycount);
    hipMalloc((void**)&d_ips, sizeof(bool) * waitscount);
    
    hipMemcpy(d_waits, waits, sizeof(Point) * waitscount, hipMemcpyHostToDevice);
    hipMemcpy(d_Polypoints, Polypoints, sizeof(Point) * Polycount, hipMemcpyHostToDevice);

    unsigned int gridlength = (waitscount + BLOCK_SIZE - 1) / BLOCK_SIZE;
    gpu_internalpoints << <gridlength, BLOCK_SIZE >> > (d_waits,d_Polypoints, Polycount,d_ips);
    hipMemcpy(containIndex, d_ips, sizeof(bool) *waitscount, hipMemcpyDeviceToHost);
    vector<Point> ips;
    for(int i=0;i<waitscount;i++){
        if(containIndex[i]==true)
        ips.push_back(waits[i]);
    }
    
    //for(int i=0;i<ips.size();i++){
    //    ips[i].print();
    //}

    writewkt(ips);

    return 0;



}



