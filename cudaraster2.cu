#include "hip/hip_runtime.h"
//#include <bits/stdc++.h>
#include <iostream>
#include <fstream>
#include <cassert>
#include <string>
#include <sstream>
#include <vector>

#include "MyPolygon.h"


using namespace std;

#define BLOCK_SIZE 16

//传二维没必要 传bool也没必要

__global__ void gpu_internalpoints(double start_x,double start_y,double step, int dimx, Point* p, int num_vertices, bool* ips)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

	double low[2],high[2];
	low[0] = row*step+start_x;
	high[0] = (row+1.0)*step+start_x;
	low[1] = col*step+start_y;
	high[1] = (col+1.0)*step+start_y;

	double x=(low[0]+high[0])/2;
	double y=(low[1]+high[1])/2;
	//temp.print();


	int Pointindex= row*blockDim.x+col;				//row*liekuan +col
	ips[Pointindex]=false;
	int i = 0, j = num_vertices - 1;
	for (i; i < num_vertices; i++) {
		if ((y >= p[i].y) != (y >= p[j].y)) {
			int linex = p[i].x + (y - p[i].y) * (p[i].x - p[j].x) / (p[i].y - p[j].y);
			if (linex > x) {
				ips[Pointindex] = !ips[Pointindex];
			}
		}
		j = i;
	}
}

string readhang(string file){
    string hang;
    ifstream infile;
    infile.open(file.data());
    assert(infile.is_open());

    getline(infile,hang);
    getline(infile,hang);
    //cout<<wkt;
    infile.close();
    return hang;
}

void stringsplit(string str,const char split,vector<string>& raw){
	istringstream iss(str);
	string token;
	while (getline(iss, token, split))
	{
		raw.push_back(token);
	}
}

void rawpointsplit(vector<string>& raw,const char split,Point *coordinate){
	string token;
	stringstream stream;
	double n;
	for(int i=0;i<raw.size();i++){
        istringstream iss(raw[i]);
        getline(iss, token, split);
        stream<<token;
        assert(n>=0);
        stream>>n;
        coordinate[i].x=n;
        stream.clear();

        getline(iss, token, split);
        stream<<token;
        stream>>n;
        assert(n>=0);
        coordinate[i].y=n;
        //coordinate[i].print();
        stream.clear();
	}

}

vector<Point > internalpoints(VertexSequence polygon,MyRaster raster){
    int dimx=raster.get_dimx();
    int dimy=raster.get_dimy();
    vector<Point > ip;
	for(double i=0;i<=dimx;i++){
		for(double j=0;j<=dimy;j++){
            Pixel *checkpix=raster.get(i,j);
            if(polygon.contain(checkpix->centralpoint)){
                ip.push_back(checkpix->centralpoint);
            }

		}
	};
	return ip;
}

void writewkt(vector<Point> ips){
    ofstream out("ips.wkt");
    if (out.is_open()){
        out << "MultiPoint(";
        for(int i=0;i<ips.size();i++){
            if(i>=1) out<<",";
            out<<"("<<ips[i].x<<" "<<ips[i].y<<")";
        }
        out<<")";
        out.close();
    }
}

int main(){
    string filename="bigpolygon.wkt";           
    string wkt=readhang(filename);
    vector<string> rawpoints;

	stringsplit(wkt,',',rawpoints);
    rawpoints[0]=rawpoints[0].substr(9,rawpoints[0].length());                      //POLYGON((129.536643 49.398036
    int last=rawpoints.size()-1;
    rawpoints[last]=rawpoints[last].substr(0,rawpoints[last].length()-2);           //129.520371 49.413440))

    Point *pp=new Point[rawpoints.size()];
    rawpointsplit(rawpoints,' ',pp);
    VertexSequence polygon(rawpoints.size(),pp);
    cout<<polygon.num_vertices<<endl;
    Point position(92.6, 43.7);
    if(polygon.contain(position)){
        cout<<"surely contain";
    }
    else cout<<"out";
    cout<<endl;
    box *MBR=polygon.getMBR();
    MBR->print();
    cout<<endl;
    double s=MBR->area();
    cout<<s<<endl;

    int num_pixel=200000;
    MyRaster raster(&polygon,num_pixel);
    //raster.init_pixels();

    double start_x, start_y, step;
    start_x=raster.get_start_x();
    start_y=raster.get_start_y();
    step=raster.get_step_x();

    int waitscount=num_pixel;			//raster.candidates.size()
    int Polycount=polygon.num_vertices;
    Point *Polypoints;
    bool *containIndex;
    //hipHostMalloc((void**)&waits, sizeof(Point) * waitscount);
    hipHostMalloc((void**)&Polypoints, sizeof(Point) * Polycount);
    hipHostMalloc((void**)&containIndex, sizeof(bool) * waitscount);
//    for(int i=0;i<waitscount;i++){
//    	waits[i]=raster.candidates.front();
//    	raster.candidates.pop();
//    }
    
    Polypoints=polygon.p;
    Point *d_Polypoints;
    bool *d_ips;
    //hipMalloc((void**)&d_waits, sizeof(Point) * waitscount);
    hipMalloc((void**)&d_Polypoints, sizeof(Point) * Polycount);
    hipMalloc((void**)&d_ips, sizeof(bool) * waitscount);
    
    //hipMemcpy(d_waits, waits, sizeof(Point) * waitscount, hipMemcpyHostToDevice);
    hipMemcpy(d_Polypoints, Polypoints, sizeof(Point) * Polycount, hipMemcpyHostToDevice);
    int dimx=raster.get_dimx();
    int dimy=raster.get_dimy();
	unsigned int grid_rows = (dimx + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int grid_cols = (dimy + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	cout<<"before Entry gpu"<<endl;
    gpu_internalpoints << <dimGrid, dimBlock >> > (start_x, start_y, step, dimx, d_Polypoints, Polycount,d_ips);
    cout<<"After gpu"<<endl;
    hipMemcpy(containIndex, d_ips, sizeof(bool) *waitscount, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    vector<Point> ips;
    
    for(double i=0;i<=dimx;i++){
		for(double j=0;j<=dimy;j++){
			int Pointindex= i*dimx+j;
			if(containIndex[Pointindex]==true){
				Pixel *m = new Pixel();

				m->id[0] = i;
				m->id[1] = j;
				m->low[0] = i*step+start_x;
				m->high[0] = (i+1.0)*step+start_x;
				m->low[1] = j*step+start_y;
				m->high[1] = (j+1.0)*step+start_y;

				m->centralpoint.x=(m->low[0]+m->high[0])/2;
				m->centralpoint.y=(m->low[1]+m->high[1])/2;
				//m->centralpoint.print();
				ips.push_back(m->centralpoint);
			}
		}
	};

    //for(int i=0;i<ips.size();i++){
    //    ips[i].print();
    //}

    writewkt(ips);

    return 0;



}



