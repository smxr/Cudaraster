#include "hip/hip_runtime.h"
//#include <bits/stdc++.h>
//#include <iostream>
#include <fstream>
#include <cassert>
#include <string>
#include <sstream>
#include <vector>

#include "MyPolygon.h"


using namespace std;

#define BLOCK_SIZE 1024

__global__ void gpu_internalpoints(Point* po, int num_pix, Point* p, int num_vertices)         //(d_waits,d_Polypoints,Polycount,d_ips)
{
    int Pointindex = blockIdx.x * blockDim.x + threadIdx.x;
    if(Pointindex<num_pix){
    	assert(po[Pointindex].x!=0&&po[Pointindex].y!=0);
    	bool temp=false;			//false=out    true=in
 		int i = 0, j = num_vertices - 1;
		for (i; i < num_vertices; i++) {
			if ((po[Pointindex].y >= p[i].y) != (po[Pointindex].y >= p[j].y)) {
				int linex = p[i].x + (po[Pointindex].y - p[i].y) * (p[i].x - p[j].x) / (p[i].y - p[j].y);
				if (linex > po[Pointindex].x) {
					temp=!temp;
				}
			}
			j = i;
		}
		if(temp==false){
			po[Pointindex].x=0;
			po[Pointindex].y=0;
		}
    }
}

string readhang(string file){
    string hang;
    ifstream infile;
    infile.open(file.data());
    assert(infile.is_open());

    getline(infile,hang);
    getline(infile,hang);
    //cout<<wkt;
    infile.close();
    return hang;
}

void stringsplit(string str,const char split,vector<string>& raw){
	istringstream iss(str);
	string token;
	while (getline(iss, token, split))
	{
		raw.push_back(token);
	}
}

void rawpointsplit(vector<string>& raw,const char split,Point *coordinate){
	string token;
	stringstream stream;
	double n;
	for(int i=0;i<raw.size();i++){
        istringstream iss(raw[i]);
        getline(iss, token, split);
        stream<<token;
        assert(n>=0);
        stream>>n;
        coordinate[i].x=n;
        stream.clear();

        getline(iss, token, split);
        stream<<token;
        stream>>n;
        assert(n>=0);
        coordinate[i].y=n;
        //coordinate[i].print();
        stream.clear();
	}

}

vector<Point > internalpoints(VertexSequence polygon,MyRaster raster){
    int dimx=raster.get_dimx();
    int dimy=raster.get_dimy();
    vector<Point > ip;
	for(double i=0;i<=dimx;i++){
		for(double j=0;j<=dimy;j++){
            Pixel *checkpix=raster.get(i,j);
            if(polygon.contain(checkpix->centralpoint)){
                ip.push_back(checkpix->centralpoint);
            }

		}
	};
	return ip;
}

void writewkt(vector<Point> ips){
    ofstream out("ips.wkt");
    if (out.is_open()){
        out << "MultiPoint(";
        for(int i=0;i<ips.size();i++){
            if(i>=1) out<<",";
            out<<"("<<ips[i].x<<" "<<ips[i].y<<")";
        }
        out<<")";
        out.close();
    }
}

int main(){
    string filename="bigpolygon.wkt";
    string wkt=readhang(filename);
    vector<string> rawpoints;

	stringsplit(wkt,',',rawpoints);
    rawpoints[0]=rawpoints[0].substr(9,rawpoints[0].length());                      //POLYGON((129.536643 49.398036
    int last=rawpoints.size()-1;
    rawpoints[last]=rawpoints[last].substr(0,rawpoints[last].length()-2);           //129.520371 49.413440))

    Point *pp=new Point[rawpoints.size()];
    rawpointsplit(rawpoints,' ',pp);
    VertexSequence polygon(rawpoints.size(),pp);
    cout<<polygon.num_vertices<<endl;
    Point position(92.6, 43.7);
    if(polygon.contain(position)){
        cout<<"surely contain";
    }
    else cout<<"out";
    cout<<endl;
    box *MBR=polygon.getMBR();
    MBR->print();
    cout<<endl;
    double s=MBR->area();
    cout<<s<<endl;

    int num_pixel=20000;
    MyRaster raster(&polygon,num_pixel);
    raster.init_pixels();

    int waitscount=raster.candidates.size();
    int Polycount=polygon.num_vertices;
    Point *waits, *Polypoints;
    hipHostMalloc((void**)&waits, sizeof(Point) * waitscount);
    hipHostMalloc((void**)&Polypoints, sizeof(Point) * Polycount);
    for(int i=0;i<waitscount;i++){
    	waits[i]=raster.candidates.front();
    	raster.candidates.pop();
    }

    Polypoints=polygon.p;
    Point *d_waits, *d_Polypoints;
    hipMalloc((void**)&d_waits, sizeof(Point) * waitscount);
    hipMalloc((void**)&d_Polypoints, sizeof(Point) * Polycount);

    hipMemcpy(d_waits, waits, sizeof(Point) * waitscount, hipMemcpyHostToDevice);
    hipMemcpy(d_Polypoints, Polypoints, sizeof(Point) * Polycount, hipMemcpyHostToDevice);

    unsigned int gridlength = (waitscount + BLOCK_SIZE - 1) / BLOCK_SIZE;
    SpendTime *time=new SpendTime();
    gpu_internalpoints << <gridlength, BLOCK_SIZE >> > (d_waits, waitscount, d_Polypoints, Polycount);
    hipDeviceSynchronize();
    cout<<"Gpu:";
    delete time;
    hipMemcpy(waits, d_waits, sizeof(Point) * waitscount, hipMemcpyDeviceToHost);
    vector<Point> ips;
    for(int i=0;i<waitscount;i++){
        if(waits[i].x==0&&waits[i].y==0);
        else ips.push_back(waits[i]);
    }


    MyRaster raster0(&polygon,num_pixel);
	raster0.init_pixels();
	SpendTime *time0=new SpendTime();
	vector<Point> ips0=internalpoints(polygon,raster0);
	cout<<"Cpu:";
	delete time0;
	cout<<"ips.size"<<ips.size()<<endl;
	cout<<"ips0.size"<<ips0.size()<<endl;
	if(ips.size()==ips0.size()){
		cout<<"size right"<<endl;
		for(int i=0;i<ips.size();i++){
			if(ips[i].x!=ips0[i].x){
				cout<<i<<"diffderence"<<endl;
				break;
			}
			if(ips[i].y!=ips0[i].y){
				cout<<i<<"diffderence"<<endl;
				break;
			}
		}
	}

    writewkt(ips);

    return 0;

}
//nvcc cudaraster4.cu MyPolygon.cpp MyRaster.cpp Pixel.cpp -o main4







