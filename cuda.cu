#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <cassert>
#include <string>
//#include <sstream>     
#include <vector>
#include <time.h>


using namespace std;

#define BLOCK_SIZE 16



__global__ void gpu_matrix_mult(int* a, int* b, int* c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;                //��ǰ�淴�Ŷ���ģ������ٷ�һ�ξ�����������     
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    //blockIdx.xΪdimGrid�ĵ�һ��������Ҳ����grid_cols��Χ�ڵ�һ��ֵ��  blockIdx.yΪgrid_rows��     
    //blockDim.y blockDim.x����block����Ӧ����ĳ�������16 ��ʾ�������16���߳�
    //threadIdx �߳����� ��������ˣ�����0��15��һ����

    int sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

void cpu_matrix_mult(int* h_a, int* h_b, int* h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h)
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

void writetxt(int* a, int row, int column, string txtname) {
    ofstream out(txtname + ".txt");
    if (out.is_open()) {

        for (int i = 0; i < row; ++i) {
            for (int j = 0; j < column; ++j) {
                out << a[i * row + j] << " ";
            }
            out << endl;
        }
        out.close();
    }
}

int main(int argc, char const* argv[])
{
    int m = 2000;
    int n = 2000;
    int k = 2000;

    int* h_a, * h_b, * h_c, * h_cc;
    hipHostMalloc((void**)&h_a, sizeof(int) * m * n);              
    hipHostMalloc((void**)&h_b, sizeof(int) * n * k);
    hipHostMalloc((void**)&h_c, sizeof(int) * m * k);
    hipHostMalloc((void**)&h_cc, sizeof(int) * m * k);
    //hipHostMalloc((void**)&h_ccc, sizeof(int) * m * k);

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 1024;
        }
    }
    writetxt(h_a, m, n, "h_a");
    writetxt(h_b, n, k, "h_b");

    clock_t start1, stop1, start2, stop2;
    start1 = clock();

    int* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, sizeof(int) * m * n);
    hipMalloc((void**)&d_b, sizeof(int) * n * k);
    hipMalloc((void**)&d_c, sizeof(int) * m * k);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;                 //���ö�άgrid �൱��m+1����16����ȡ��
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);                                         //�����˾�����ô�����ŵģ���ʾ��һ��grid��x y����Ĵ�С    //dim3��3ά����˼�������������Ĭ��Ϊ1���õ��Ƕ�ά
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_matrix_mult << <dimGrid, dimBlock >> > (d_a, d_b, d_c, m, n, k);     //gpu_matrix_mult����һ����һ���̴߳�������е�һ���գ��������� dimGrid*diBlock=grid_cols*grid_rows*BLOCK_SIZE*BLOCK_SIZE��>=m*k��

    hipMemcpy(h_c, d_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();

    stop1 = clock();

    start2 = clock();
    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);
    stop2 = clock();

    

    double endtime1 = (double)(stop1 - start1) / CLOCKS_PER_SEC;
    std::cout << "cudatime: " << endtime1 << "s" << std::endl;
    double endtime2 = (double)(stop2 - start2) / CLOCKS_PER_SEC;
    std::cout << "cputime: " << endtime2 << "s" << std::endl;



    int ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            if (fabs(h_cc[i * k + j] - h_c[i * k + j]) > (1.0e-10))
            {

                ok = 0;
            }
        }
    }



    if (ok )
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }

    writetxt(h_c, m, k, "h_c");
    writetxt(h_cc, m, k, "h_cc");


    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}
